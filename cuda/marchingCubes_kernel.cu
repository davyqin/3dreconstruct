#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#ifndef _MARCHING_CUBES_KERNEL_CU_
#define _MARCHING_CUBES_KERNEL_CU_

#include <stdio.h>
#include <string.h>
//#include <hip/hip_runtime_api.h>    // includes for helper CUDA functions
//#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <cmath>

#include <mc/common.h>

#include "cuda_common.h"
#include "hip/hip_vector_types.h"
#include "marching_cubes_kernel.h"

namespace {
  uint3 gridSizeShift;
  uint3 gridSize;
  uint3 gridSizeMask;

  float3 voxelSize;
  uint numVoxels = 0;
  uint maxVerts = 0;
  uint activeVoxels = 0;
  uint totalVerts = 0;

  float minIso = 0.102f;
  float maxIso = 0.902f;
  float3 startPos;

  float *d_xResult = 0;
  float *d_yResult = 0;
  float *d_zResult = 0;

  uchar *d_volume = 0;
  uint *d_voxelVerts = 0;
  uint *d_voxelVertsScan = 0;
  uint *d_voxelOccupied = 0;
  uint *d_voxelOccupiedScan = 0;
  uint *d_compVoxelArray;

  // tables
  int *d_numVertsTable = 0;
  short *d_edgeTable = 0;
  int *d_triTable = 0;


  // textures containing look-up tables
  texture<short, 1, hipReadModeElementType> edgeTex;
  texture<int, 1, hipReadModeElementType> triTex;
  texture<int, 1, hipReadModeElementType> numVertsTex;

  // volume data
  texture<unsigned char, 1, hipReadModeNormalizedFloat> volumeTex;
}

extern "C"
void allocateTextures()
{
    checkCudaErrors(hipMalloc((void **)&d_edgeTable, 256*sizeof(short)));
    checkCudaErrors(hipMemcpy((void *)d_edgeTable, (void *)EdgeTable, 256*sizeof(short), hipMemcpyHostToDevice));
    hipChannelFormatDesc channelShortDesc = hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindSigned);
    checkCudaErrors(hipBindTexture(0, edgeTex, d_edgeTable, channelShortDesc));

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindSigned);
    checkCudaErrors(hipMalloc((void **)&d_triTable, 256*16*sizeof(int)));
    checkCudaErrors(hipMemcpy((void *)d_triTable, (void *)TriangleTable, 256*16*sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipBindTexture(0, triTex, d_triTable, channelDesc));

    checkCudaErrors(hipMalloc((void **)&d_numVertsTable, 256*sizeof(int)));
    checkCudaErrors(hipMemcpy((void *)d_numVertsTable, (void *)numVertsTable, 256*sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipBindTexture(0, numVertsTex, d_numVertsTable, channelDesc));
}

// sample volume data set at a point
__device__
float sampleVolume(uchar *data, uint3 p, uint3 gridSize)
{
    p.x = min(p.x, gridSize.x - 1);
    p.y = min(p.y, gridSize.y - 1);
    p.z = min(p.z, gridSize.z - 1);
    uint i = (p.z*gridSize.x*gridSize.y) + (p.y*gridSize.x) + p.x;
    //    return (float) data[i] / 255.0f;
    return tex1Dfetch(volumeTex, i);
}

// compute position in 3d grid from 1d index
// only works for power of 2 sizes
__device__
uint3 calcGridPos(uint i, uint3 gridSizeShift, uint3 gridSizeMask)
{
    uint3 gridPos;
    gridPos.x = i & gridSizeMask.x;
    gridPos.y = (i >> gridSizeShift.y) & gridSizeMask.y;
    gridPos.z = (i >> gridSizeShift.z) & gridSizeMask.z;
    return gridPos;
}

// classify voxel based on number of vertices it will generate
// one thread per voxel
__global__ void
classifyVoxel(uint *voxelVerts, uint *voxelOccupied, uchar *volume,
              uint3 gridSize, uint3 gridSizeShift, uint3 gridSizeMask, uint numVoxels,
              float3 voxelSize, float minIso, float maxIso)
{
    uint blockId = __mul24(blockIdx.y, gridDim.x) + blockIdx.x;
    uint i = __mul24(blockId, blockDim.x) + threadIdx.x;

    uint3 gridPos = calcGridPos(i, gridSizeShift, gridSizeMask);
    __syncthreads();

    // read field values at neighbouring grid vertices
    float field[8];
    field[0] = sampleVolume(volume, gridPos, gridSize);
    field[1] = sampleVolume(volume, gridPos + make_uint3(1, 0, 0), gridSize);
    field[2] = sampleVolume(volume, gridPos + make_uint3(1, 1, 0), gridSize);
    field[3] = sampleVolume(volume, gridPos + make_uint3(0, 1, 0), gridSize);
    field[4] = sampleVolume(volume, gridPos + make_uint3(0, 0, 1), gridSize);
    field[5] = sampleVolume(volume, gridPos + make_uint3(1, 0, 1), gridSize);
    field[6] = sampleVolume(volume, gridPos + make_uint3(1, 1, 1), gridSize);
    field[7] = sampleVolume(volume, gridPos + make_uint3(0, 1, 1), gridSize);

    // calculate flag indicating if each vertex is inside or outside isosurface
    uint cubeindex;
    cubeindex =  uint(field[0] >= minIso && field[0] <= maxIso);
    cubeindex += uint(field[1] >= minIso && field[1] <= maxIso)*2;
    cubeindex += uint(field[2] >= minIso && field[2] <= maxIso)*4;
    cubeindex += uint(field[3] >= minIso && field[3] <= maxIso)*8;
    cubeindex += uint(field[4] >= minIso && field[4] <= maxIso)*16;
    cubeindex += uint(field[5] >= minIso && field[5] <= maxIso)*32;
    cubeindex += uint(field[6] >= minIso && field[6] <= maxIso)*64;
    cubeindex += uint(field[7] >= minIso && field[7] <= maxIso)*128;

    __syncthreads();

    // read number of vertices from texture
    uint numVerts = tex1Dfetch(numVertsTex, cubeindex);

    if (i < numVoxels)
    {
        voxelVerts[i] = numVerts;
        voxelOccupied[i] = (numVerts > 0);
    }

    __syncthreads();
}

extern "C" void
launch_classifyVoxel(dim3 grid, dim3 threads, uint *voxelVerts, uint *voxelOccupied, uchar *volume,
                     uint3 gridSize, uint3 gridSizeShift, uint3 gridSizeMask, uint numVoxels,
                     float3 voxelSize)
{
    // calculate number of vertices need per voxel
    classifyVoxel<<<grid, threads>>>(voxelVerts, voxelOccupied, volume,
                                     gridSize, gridSizeShift, gridSizeMask,
                                     numVoxels, voxelSize, minIso, maxIso);
    getLastCudaError("classifyVoxel failed");
}

// compact voxel array
__global__ void
compactVoxels(uint *compactedVoxelArray, uint *voxelOccupied, uint *voxelOccupiedScan, uint numVoxels)
{
    uint blockId = __mul24(blockIdx.y, gridDim.x) + blockIdx.x;
    uint i = __mul24(blockId, blockDim.x) + threadIdx.x;

    if (voxelOccupied[i] && (i < numVoxels))
    {
        compactedVoxelArray[ voxelOccupiedScan[i] ] = i;
    }
}

extern "C" void
launch_compactVoxels(dim3 grid, dim3 threads, uint *compactedVoxelArray, uint *voxelOccupied, uint *voxelOccupiedScan, uint numVoxels)
{
    compactVoxels<<<grid, threads>>>(compactedVoxelArray, voxelOccupied,
                                     voxelOccupiedScan, numVoxels);
    getLastCudaError("compactVoxels failed");
}

// compute interpolated vertex along an edge
__device__
float3 vertexInterp(float minIso, float maxIso, float3 p0, float3 p1, float f0, float f1)
{
    float dv = f0 - f1;
    float dValMax1 = f0 - maxIso;
    float dMaxVal2 = maxIso - f1;
    float dValMin1 = f0 - minIso;
    float dMinVal2 = minIso - f1;

    if (f0 >= minIso && f0 <= maxIso) {
      if (f1 > maxIso) {
        return (p1 * dValMax1 + p0 * dMaxVal2) / dv;
      }
      else {
        return (p1 * dValMin1 + p0 * dMinVal2) / dv;
      }
    }
    else if (f0 > maxIso) {
      return (p1 * dValMax1 + p0 * dMaxVal2) / dv;
    }
    else {
      return (p1 * dValMin1 + p0 * dMinVal2) / dv;
    }
}

// version that calculates flat surface normal for each triangle
__global__ void
generateTriangles2(uint *numVertsScanned, uint *compactedVoxelArray, uchar *volume, uint3 gridSize, uint3 gridSizeShift, uint3 gridSizeMask,
                   float3 voxelSize, float minIso, float maxIso, uint activeVoxels, uint maxVerts, float3 startPos,
                   float* xPos, float* yPos, float* zPos)
{
    uint blockId = __mul24(blockIdx.y, gridDim.x) + blockIdx.x;
    uint i = __mul24(blockId, blockDim.x) + threadIdx.x;

    if (i > activeVoxels - 1)
    {
        i = activeVoxels - 1;
    }

#if SKIP_EMPTY_VOXELS
    uint voxel = compactedVoxelArray[i];
#else
    uint voxel = i;
#endif

    uint3 gridPos = calcGridPos(voxel, gridSizeShift, gridSizeMask);

    float3 p;
    p.x = startPos.x + (gridPos.x * voxelSize.x);
    p.y = startPos.y + (gridPos.y * voxelSize.y);
    p.z = startPos.z + (gridPos.z * voxelSize.z);

    // calculate cell vertex positions
    float3 v[8];
    v[0] = p;
    v[1] = p + make_float3(voxelSize.x, 0, 0);
    v[2] = p + make_float3(voxelSize.x, voxelSize.y, 0);
    v[3] = p + make_float3(0, voxelSize.y, 0);
    v[4] = p + make_float3(0, 0, voxelSize.z);
    v[5] = p + make_float3(voxelSize.x, 0, voxelSize.z);
    v[6] = p + make_float3(voxelSize.x, voxelSize.y, voxelSize.z);
    v[7] = p + make_float3(0, voxelSize.y, voxelSize.z);

    float field[8];
    field[0] = sampleVolume(volume, gridPos, gridSize);
    field[1] = sampleVolume(volume, gridPos + make_uint3(1, 0, 0), gridSize);
    field[2] = sampleVolume(volume, gridPos + make_uint3(1, 1, 0), gridSize);
    field[3] = sampleVolume(volume, gridPos + make_uint3(0, 1, 0), gridSize);
    field[4] = sampleVolume(volume, gridPos + make_uint3(0, 0, 1), gridSize);
    field[5] = sampleVolume(volume, gridPos + make_uint3(1, 0, 1), gridSize);
    field[6] = sampleVolume(volume, gridPos + make_uint3(1, 1, 1), gridSize);
    field[7] = sampleVolume(volume, gridPos + make_uint3(0, 1, 1), gridSize);

    // recalculate flag
    uint cubeindex;
    cubeindex =  uint(field[0] >= minIso && field[0] <= maxIso);
    cubeindex += uint(field[1] >= minIso && field[1] <= maxIso)*2;
    cubeindex += uint(field[2] >= minIso && field[2] <= maxIso)*4;
    cubeindex += uint(field[3] >= minIso && field[3] <= maxIso)*8;
    cubeindex += uint(field[4] >= minIso && field[4] <= maxIso)*16;
    cubeindex += uint(field[5] >= minIso && field[5] <= maxIso)*32;
    cubeindex += uint(field[6] >= minIso && field[6] <= maxIso)*64;
    cubeindex += uint(field[7] >= minIso && field[7] <= maxIso)*128;

 #if USE_SHARED
    // use shared memory to avoid using local
    __shared__ float3 vertlist[12*NTHREADS];

    vertlist[threadIdx.x] = vertexInterp(minIso, maxIso, v[0], v[1], field[0], field[1]);
    vertlist[NTHREADS+threadIdx.x] = vertexInterp(minIso, maxIso, v[1], v[2], field[1], field[2]);
    vertlist[(NTHREADS*2)+threadIdx.x] = vertexInterp(minIso, maxIso, v[2], v[3], field[2], field[3]);
    vertlist[(NTHREADS*3)+threadIdx.x] = vertexInterp(minIso, maxIso, v[3], v[0], field[3], field[0]);
    vertlist[(NTHREADS*4)+threadIdx.x] = vertexInterp(minIso, maxIso, v[4], v[5], field[4], field[5]);
    vertlist[(NTHREADS*5)+threadIdx.x] = vertexInterp(minIso, maxIso, v[5], v[6], field[5], field[6]);
    vertlist[(NTHREADS*6)+threadIdx.x] = vertexInterp(minIso, maxIso, v[6], v[7], field[6], field[7]);
    vertlist[(NTHREADS*7)+threadIdx.x] = vertexInterp(minIso, maxIso, v[7], v[4], field[7], field[4]);
    vertlist[(NTHREADS*8)+threadIdx.x] = vertexInterp(minIso, maxIso, v[0], v[4], field[0], field[4]);
    vertlist[(NTHREADS*9)+threadIdx.x] = vertexInterp(minIso, maxIso, v[1], v[5], field[1], field[5]);
    vertlist[(NTHREADS*10)+threadIdx.x] = vertexInterp(minIso, maxIso, v[2], v[6], field[2], field[6]);
    vertlist[(NTHREADS*11)+threadIdx.x] = vertexInterp(minIso, maxIso, v[3], v[7], field[3], field[7]);
    __syncthreads();

#else

    // find the vertices where the surface intersects the cube
    float3 vertlist[12];

    vertlist[0] = vertexInterp(minIso, maxIso, v[0], v[1], field[0], field[1]);
    vertlist[1] = vertexInterp(minIso, maxIso, v[1], v[2], field[1], field[2]);
    vertlist[2] = vertexInterp(minIso, maxIso, v[2], v[3], field[2], field[3]);
    vertlist[3] = vertexInterp(minIso, maxIso, v[3], v[0], field[3], field[0]);

    vertlist[4] = vertexInterp(minIso, maxIso, v[4], v[5], field[4], field[5]);
    vertlist[5] = vertexInterp(minIso, maxIso, v[5], v[6], field[5], field[6]);
    vertlist[6] = vertexInterp(minIso, maxIso, v[6], v[7], field[6], field[7]);
    vertlist[7] = vertexInterp(minIso, maxIso, v[7], v[4], field[7], field[4]);

    vertlist[8] = vertexInterp(minIso, maxIso, v[0], v[4], field[0], field[4]);
    vertlist[9] = vertexInterp(minIso, maxIso, v[1], v[5], field[1], field[5]);
    vertlist[10] = vertexInterp(minIso, maxIso, v[2], v[6], field[2], field[6]);
    vertlist[11] = vertexInterp(minIso, maxIso, v[3], v[7], field[3], field[7]);
#endif

    // output triangle vertices
    uint numVerts = tex1Dfetch(numVertsTex, cubeindex);

    for (int i = 0; i < numVerts; i+=3)
    {
        uint index = numVertsScanned[voxel] + i;

        float3 v[3];
        int edge;
        edge = tex1Dfetch(triTex, (cubeindex*16) + i);
#if USE_SHARED
        v[0] = vertlist[(edge*NTHREADS)+threadIdx.x];
#else
        v[0] = vertlist[edge];
#endif

        edge = tex1Dfetch(triTex, (cubeindex*16) + i + 1);
#if USE_SHARED
        v[1] = vertlist[(edge*NTHREADS)+threadIdx.x];
#else
        v[1] = vertlist[edge];
#endif


        edge = tex1Dfetch(triTex, (cubeindex*16) + i + 2);
#if USE_SHARED
        v[2] = vertlist[(edge*NTHREADS)+threadIdx.x];
#else
        v[2] = vertlist[edge];
#endif

        if (index < (maxVerts - 3))
        {
          xPos[index] = v[0].x;
          yPos[index] = v[0].y;
          zPos[index] = v[0].z;

          xPos[index+1] = v[1].x;
          yPos[index+1] = v[1].y;
          zPos[index+1] = v[1].z;

          xPos[index+2] = v[2].x;
          yPos[index+2] = v[2].y;
          zPos[index+2] = v[2].z;
        }
    }
}

extern "C" void
launch_generateTriangles2(dim3 grid, dim3 threads, uint *numVertsScanned, uint *compactedVoxelArray, uchar *volume,
                          uint3 gridSize, uint3 gridSizeShift, uint3 gridSizeMask,
                          float3 voxelSize, uint activeVoxels, uint maxVerts)
{
    generateTriangles2<<<grid, NTHREADS>>>(numVertsScanned, compactedVoxelArray, volume,
                                           gridSize, gridSizeShift, gridSizeMask,
                                           voxelSize, minIso, maxIso, activeVoxels,
                                           maxVerts, startPos, d_xResult, d_yResult, d_zResult);
    getLastCudaError("generateTriangles2 failed");
}

extern "C" void ThrustScanWrapper(unsigned int *output, unsigned int *input, unsigned int numElements)
{
  thrust::exclusive_scan(thrust::device_ptr<unsigned int>(input),
                         thrust::device_ptr<unsigned int>(input + numElements),
                         thrust::device_ptr<unsigned int>(output));
}

extern "C" void initMC(int min, int max, int xyValue, int zValue, float xSpaceing, float ySpacing, float zSpacing)
{
  allocateTextures();

  minIso = (float)(min)/255.0f;
  maxIso = (float)(max)/255.0f;
  gridSize = make_uint3(xyValue, xyValue, zValue);
  gridSizeMask = make_uint3(gridSize.x-1, gridSize.y-1, gridSize.z-1);
  const int logBase2 = log10(xyValue) / log10(2);
  gridSizeShift = make_uint3(0, logBase2, logBase2*2);

  numVoxels = gridSize.x*gridSize.y*(gridSize.z - 1);
  voxelSize = make_float3(xSpaceing, ySpacing, zSpacing);
  maxVerts = gridSize.x * gridSize.y * 50;

  int size = gridSize.x*gridSize.y*gridSize.z*sizeof(uchar);
  checkCudaErrors(hipMalloc((void **) &d_volume, size));

  // allocate device memory
  unsigned int memSize = sizeof(uint) * numVoxels;
  checkCudaErrors(hipMalloc((void **) &d_voxelVerts,            memSize));
  checkCudaErrors(hipMalloc((void **) &d_voxelVertsScan,        memSize));
  checkCudaErrors(hipMalloc((void **) &d_voxelOccupied,         memSize));
  checkCudaErrors(hipMalloc((void **) &d_voxelOccupiedScan,     memSize));
  checkCudaErrors(hipMalloc((void **) &d_compVoxelArray,        memSize));
  checkCudaErrors(hipMalloc((void **) &(d_xResult),             maxVerts*sizeof(float)));
  checkCudaErrors(hipMalloc((void **) &(d_yResult),             maxVerts*sizeof(float)));
  checkCudaErrors(hipMalloc((void **) &(d_zResult),             maxVerts*sizeof(float)));
}

extern "C"
void bindVolumeTexture(unsigned char *volume)
{
  int size = gridSize.x*gridSize.y*gridSize.z*sizeof(uchar);
  checkCudaErrors(hipMemcpy(d_volume, volume, size, hipMemcpyHostToDevice));
  
  // bind to linear texture
  checkCudaErrors(hipBindTexture(0, volumeTex, d_volume, hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned)));
}

extern "C" void computeIsosurface(unsigned char *volume, float x, float y, float z, float* xResult, float* yResult, float* zResult, uint& totalVertices)
{
  bindVolumeTexture(volume);

  startPos.x = x;
  startPos.y = y;
  startPos.z = z;

  int threads = 128;
  dim3 grid(numVoxels / threads, 1, 1);

 // get around maximum grid size of 65535 in each dimension
  if (grid.x > 65535)
  {
    grid.y = grid.x / 32768;
    grid.x = 32768;
  }

  // calculate number of vertices need per voxel
  launch_classifyVoxel(grid, threads,
                       d_voxelVerts, d_voxelOccupied, d_volume,
                       gridSize, gridSizeShift, gridSizeMask,
                       numVoxels, voxelSize);

#if SKIP_EMPTY_VOXELS
#if _DEBUG
    {
    uint* voxelOccupied = new uint[numVoxels];
    checkCudaErrors(hipMemcpy(voxelOccupied, d_voxelOccupied, numVoxels * sizeof(uint), hipMemcpyDeviceToHost));
  
    uint* voxelOccupiedScan = new uint[numVoxels+1];
    voxelOccupiedScan[0] = 0;
    for (int i = 1; i <= numVoxels; ++i) {
      voxelOccupiedScan[i] = voxelOccupiedScan[i - 1] + voxelOccupied[i -1];
    }

    activeVoxels = voxelOccupiedScan[numVoxels];
    checkCudaErrors(hipMemcpy(d_voxelOccupiedScan, voxelOccupiedScan, numVoxels * sizeof(uint), hipMemcpyHostToDevice));

    delete [] voxelOccupied;
    delete [] voxelOccupiedScan;
  }
#else

    // scan voxel occupied array
    ThrustScanWrapper(d_voxelOccupiedScan, d_voxelOccupied, numVoxels);

    // read back values to calculate total number of non-empty voxels
    // since we are using an exclusive scan, the total is the last value of
    // the scan result plus the last value in the input array
    {
        uint lastElement, lastScanElement;
        checkCudaErrors(hipMemcpy((void *) &lastElement,
                                   (void *)(d_voxelOccupied + numVoxels-1),
                                   sizeof(uint), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy((void *) &lastScanElement,
                                   (void *)(d_voxelOccupiedScan + numVoxels-1),
                                   sizeof(uint), hipMemcpyDeviceToHost));
        activeVoxels = lastElement + lastScanElement;
    }
#endif
    if (activeVoxels==0)
    {
        // return if there are no full voxels
        totalVerts = 0;
        return;
    }

    // compact voxel index array
    launch_compactVoxels(grid, threads, d_compVoxelArray, d_voxelOccupied, d_voxelOccupiedScan, numVoxels);
    getLastCudaError("compactVoxels failed");

#endif // SKIP_EMPTY_VOXELS

#if _DEBUG
  
  uint* voxelVerts = new uint[numVoxels];
  checkCudaErrors(hipMemcpy(voxelVerts, d_voxelVerts, numVoxels * sizeof(uint), hipMemcpyDeviceToHost));
  
  uint* voxelVertsScan = new uint[numVoxels+1];
  voxelVertsScan[0] = 0;
  for (int i = 1; i <= numVoxels; ++i) {
    voxelVertsScan[i] = voxelVertsScan[i - 1] + voxelVerts[i -1];
  }

  totalVerts = voxelVertsScan[numVoxels];
  totalVertices = totalVerts;
  checkCudaErrors(hipMemcpy(d_voxelVertsScan, voxelVertsScan, numVoxels * sizeof(uint), hipMemcpyHostToDevice));

  delete [] voxelVerts;
  delete [] voxelVertsScan;
  
#else
   // scan voxel vertex count array
  ThrustScanWrapper(d_voxelVertsScan, d_voxelVerts, numVoxels);

  // readback total number of vertices
  {
      uint lastElement, lastScanElement;
      checkCudaErrors(hipMemcpy((void *) &lastElement,
                                  (void *)(d_voxelVerts + numVoxels-1),
                                  sizeof(uint), hipMemcpyDeviceToHost));
      checkCudaErrors(hipMemcpy((void *) &lastScanElement,
                                  (void *)(d_voxelVertsScan + numVoxels-1),
                                  sizeof(uint), hipMemcpyDeviceToHost));
      totalVerts = lastElement + lastScanElement;
      totalVertices = totalVerts;
  }
#endif

  dim3 grid2((int) ceil(numVoxels / (float) NTHREADS), 1, 1);

  while (grid2.x > 65535)
  {
    grid2.x/=2;
    grid2.y*=2;
  }

  launch_generateTriangles2(grid2, NTHREADS, d_voxelVertsScan, d_compVoxelArray, d_volume,
                            gridSize, gridSizeShift, gridSizeMask,
                            voxelSize, activeVoxels, maxVerts);

  checkCudaErrors(hipMemcpy(xResult, d_xResult, maxVerts * sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(yResult, d_yResult, maxVerts * sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(zResult, d_zResult, maxVerts * sizeof(float), hipMemcpyDeviceToHost));
}

extern "C" void cleanup()
{
  checkCudaErrors(hipFree(d_edgeTable));
  checkCudaErrors(hipFree(d_triTable));
  checkCudaErrors(hipFree(d_numVertsTable));

  checkCudaErrors(hipFree(d_voxelVerts));
  checkCudaErrors(hipFree(d_voxelVertsScan));
  checkCudaErrors(hipFree(d_voxelOccupied));
  checkCudaErrors(hipFree(d_voxelOccupiedScan));
  checkCudaErrors(hipFree(d_compVoxelArray));
  checkCudaErrors(hipFree(d_xResult));
  checkCudaErrors(hipFree(d_yResult));
  checkCudaErrors(hipFree(d_zResult));

  if (d_volume)
  {
    checkCudaErrors(hipFree(d_volume));
  }
}

#endif
