#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>

#include "cuda_kernels.h"
#include "cuda_common.h"

// Texture reference for reading image
texture<unsigned char, 2> tex;
extern __shared__ unsigned char LocalBlock[];
static hipArray *array = NULL;

__device__ unsigned char
ComputeSobel(unsigned char ul, // upper left
             unsigned char um, // upper middle
             unsigned char ur, // upper right
             unsigned char ml, // middle left
             unsigned char mm, // middle (unused)
             unsigned char mr, // middle right
             unsigned char ll, // lower left
             unsigned char lm, // lower middle
             unsigned char lr, // lower right
             float fScale)
{
    short Horz = ur + 2*mr + lr - ul - 2*ml - ll;
    short Vert = ul + 2*um + ur - ll - 2*lm - lr;
    short Sum = (short)(fScale*(abs((int)Horz)+abs((int)Vert)));

    if (Sum < 0)
    {
        return 0;
    }
    else if (Sum > 0xff)
    {
        return 0xff;
    }

    return (unsigned char) Sum;
}

__global__ void
SobelTex(Pixel *pSobelOriginal, unsigned int Pitch, int w, int h)
{
    unsigned char *pSobel =
        (unsigned char *)(((char *) pSobelOriginal)+blockIdx.x*Pitch);

    for (int i = threadIdx.x; i < w; i += blockDim.x)
    {
        unsigned char pix00 = tex2D(tex, (float) i-1, (float) blockIdx.x-1);
        unsigned char pix01 = tex2D(tex, (float) i+0, (float) blockIdx.x-1);
        unsigned char pix02 = tex2D(tex, (float) i+1, (float) blockIdx.x-1);
        unsigned char pix10 = tex2D(tex, (float) i-1, (float) blockIdx.x+0);
        unsigned char pix11 = tex2D(tex, (float) i+0, (float) blockIdx.x+0);
        unsigned char pix12 = tex2D(tex, (float) i+1, (float) blockIdx.x+0);
        unsigned char pix20 = tex2D(tex, (float) i-1, (float) blockIdx.x+1);
        unsigned char pix21 = tex2D(tex, (float) i+0, (float) blockIdx.x+1);
        unsigned char pix22 = tex2D(tex, (float) i+1, (float) blockIdx.x+1);
        pSobel[i] = ComputeSobel(pix00, pix01, pix02,
                                 pix10, pix11, pix12,
                                 pix20, pix21, pix22, 1.0f);
    }
}

extern "C" void setupTexture(int iw, int ih, Pixel *data)
{
  hipChannelFormatDesc desc = hipCreateChannelDesc<unsigned char>();

  checkCudaErrors(hipMallocArray(&array, &desc, iw, ih));
  checkCudaErrors(hipMemcpyToArray(array, 0, 0, data, sizeof(Pixel)*iw*ih, hipMemcpyHostToDevice));
}

// Wrapper for the __global__ call that sets up the texture and threads
extern "C" void sobelFilter(Pixel *odata, int iw, int ih)
{
  //hipChannelFormatDesc desc = hipCreateChannelDesc<unsigned char>();
  checkCudaErrors(hipBindTextureToArray(tex, array));

  SobelTex<<<ih, 384>>>(odata, iw, iw, ih);

  checkCudaErrors(hipUnbindTexture(tex));
}
